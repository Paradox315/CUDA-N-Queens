
#include <hip/hip_runtime.h>
#include <stdio.h>

#define QUEENS 10

__device__
void kiir(char *A, int *db)
{
    char s[QUEENS*21 + 1];
    int k = 0;

    for(int i = 0; i < QUEENS; i++)
    {
        for(int j = 0; j < QUEENS; j++)
        {
            if(A[i] == j)
                s[k++] =  'Q';
            else
                s[k++] = '.';

            s[k++] = ' ';
        }
        s[k++] = '\n';
    }

    s[k] = '\0';

    atomicAdd(db, 1);
    printf("%d.\n%s\n", *db, s);
}

__global__
void queen(int *db, const int n)
{
    char A[QUEENS];
    
    __syncthreads();

    A[0] = threadIdx.x;
    A[1] = threadIdx.y;
    A[2] = threadIdx.z;
    A[3] = blockIdx.x/10;
    A[4] = blockIdx.x%10;
    A[5] = blockIdx.y/10;
    A[6] = blockIdx.y%10;
    A[7] = blockIdx.z/10;
    A[8] = blockIdx.z%10;
    A[9] = n;
    
    {
	bool B[QUEENS];
	
	for(int i = 0; i < QUEENS; i++)
	    B[i] = 0;
	
	for(int i = 0; i < QUEENS; i++)
	    if(!B[A[i]])
		B[A[i]] = 1;
	    else
		return;
    }

    for(int i = 0; i < QUEENS - 1; i++)
        for(int j = i + 1; j < QUEENS; j++)
            if(abs(i - j) == abs(A[i] - A[j]))
                return;
    
    kiir(A, db);
}

int
main()
{
    int h = 0, *d;
    hipMalloc((void**) &d, sizeof(int));
    hipMemcpy(d, &h, sizeof(int), hipMemcpyHostToDevice);

    dim3 blocksPerGrid(100, 100, 100);
    dim3 threadsPerBlock(10, 10, 10);
    
    for(int i = 0; i < QUEENS; i++)
	queen<<<blocksPerGrid, threadsPerBlock>>>(d, i);

    hipMemcpy(&h, d, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d);
    hipDeviceReset();

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }

    printf("Solutions: %d\n", h);

    fprintf(stderr, "\nDone\n");
    return 0;
}
