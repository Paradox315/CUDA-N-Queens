
#include <hip/hip_runtime.h>
#include <stdio.h>

#define QUEENS 10

__device__
void kiir(char *A)
{
    char s[QUEENS*21 + 1];
    int k = 0;

    for(int i = 0; i < QUEENS; i++)
    {
        for(int j = 0; j < QUEENS; j++)
        {
            if(A[i] == j)
                s[k++] =  'Q';
            else
                s[k++] = '.';

            s[k++] = ' ';
        }
        s[k++] = '\n';
    }

    s[k] = '\0';

    printf("%s\n", s);
}

__global__
void queen(int *db, const int n)
{
    {
	bool B[QUEENS];
	
	for(int i = 0; i < QUEENS; i++)
	    B[i] = 0;
	
	B[threadIdx.x] = 1;
	B[threadIdx.y] = 1;
	B[threadIdx.z] = 1;
	B[blockIdx.x/10] = 1;
	B[blockIdx.x%10] = 1;
	B[blockIdx.y/10] = 1;
	B[blockIdx.y%10] = 1;
	B[blockIdx.z/10] = 1;
	B[blockIdx.z%10] = 1;
	B[n] = 1;
	
	for(int i = 0; i < QUEENS; i++)
	    if(B[i] == 0)
		return;
    }
    
    char A[QUEENS];
    
    A[0] = threadIdx.x;
    A[1] = threadIdx.y;
    A[2] = threadIdx.z;
    A[3] = blockIdx.x/10;
    A[4] = blockIdx.x%10;
    A[5] = blockIdx.y/10;
    A[6] = blockIdx.y%10;
    A[7] = blockIdx.z/10;
    A[8] = blockIdx.z%10;
    A[9] = n;

    for(int i = 0; i < QUEENS - 1; i++)
        for(int j = i + 1; j < QUEENS; j++)
            if(abs(i - j) == abs(A[i] - A[j]))
                return;
	    
    atomicAdd(db, 1);
    printf("%d.\n", *db);
    kiir(A);
}

int
main()
{
    int h = 0, *d;
    hipMalloc((void**) &d, sizeof(int));
    hipMemcpy(d, &h, sizeof(int), hipMemcpyHostToDevice);

    dim3 blocksPerGrid(100, 100, 100);
    dim3 threadsPerBlock(10, 10, 10);
    
    for(int i = 0; i < QUEENS; i++)
	queen<<<blocksPerGrid, threadsPerBlock>>>(d, i);

    hipMemcpy(&h, d, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d);
    hipDeviceReset();

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }

    fprintf(stderr, "Solutions: %d\n", h);

    fprintf(stderr, "\nDone\n");
    return 0;
}
