#include "hip/hip_runtime.h"
#include <stdio.h>

#define QUEENS 10

__global__
void queen(int *db, const int n)
{
    int A[QUEENS];

    A[0] = threadIdx.x;
    A[1] = threadIdx.y;
    A[2] = threadIdx.z;
    A[3] = blockIdx.x/10;
    A[4] = blockIdx.x%10;
    A[5] = blockIdx.y/10;
    A[6] = blockIdx.y%10;
    A[7] = blockIdx.z/10;
    A[8] = blockIdx.z%10;
    A[9] = n;

    for(int i = 0; i < QUEENS; i++)
        for(int j = 0; j < QUEENS; j++)
            if((i != j) && (A[i] == A[j] || abs(i - j) == abs(A[i] - A[j])))
                return;

    char s[256];
    int k = 0;

    for(int i = 0; i < QUEENS; i++)
    {
        for(int j = 0; j < QUEENS; j++)
        {
            if(A[i] == j)
                s[k++] =  'Q';
            else
                s[k++] = '.';

            s[k++] = ' ';
        }
        s[k++] = '\n';
    }

    s[k] = '\0';

    atomicAdd(db, 1);
    printf("%d.\n%s\n", *db, s);
}

int
main()
{
    int h = 0, *d;
    hipMalloc((void**) &d, sizeof(int));
    hipMemcpy(d, &h, sizeof(int), hipMemcpyHostToDevice);

    dim3 blocksPerGrid(100, 100, 100);
    dim3 threadsPerBlock(10, 10, 10);

    for(int i = 0; i < QUEENS; i++)
    {
        queen<<<blocksPerGrid, threadsPerBlock>>>(d, i);
        hipDeviceSynchronize();
    }

    hipMemcpy(&h, d, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d);
    hipDeviceReset();

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }

    printf("Solutions: %d\n", h);

    fprintf(stderr, "\nDone\n");
    return 0;
}
